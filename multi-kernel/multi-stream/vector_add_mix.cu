#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#define gpuErrchk(ans) { gpuAssert( (ans), __FILE__, __LINE__ ); }

inline void
gpuAssert( hipError_t code, const char * file, int line, bool abort = true )
{
        if ( hipSuccess != code )
        {
                fprintf( stderr, "\nGPUassert: %s %s %d\n", hipGetErrorString( code ), file, line );
                if ( abort )
                        exit( code );
        }

return;

} /* gpuAssert */

__global__ void Add( int N ,int Offset ,float * devA , float * devB , float *devC )
{

        for ( int idx = blockIdx.x * blockDim.x + threadIdx.x + Offset; idx < N; idx += blockDim.x * gridDim.x )

                devC[ idx ] = devA[ idx ] + devB[ idx ];

}

int main() {

        size_t free_t, total_t;

        gpuErrchk( hipMemGetInfo(&free_t, &total_t) );
        
        //int N = 4000000;
        //unsigned long N = 1395864368;  //c.a. 1.3 * 4 GB, tot 15.6 GB
        //unsigned long N = 1288490184;  //c.a. 1.2 * 4 GB, tot 14.4 GB
        unsigned long N = 0.31 * free_t;
        
        printf("allocating %lu bytes per array\n", N);
        
        int Threads = 1024;

        const int NbStreams = 8;
        
        /************************************
                  HOST ALLOCATION
        ************************************/
        float *A , *B , *C1, *C2;
        gpuErrchk( hipHostAlloc( (void**) &A , N * sizeof(*A) ,hipHostMallocDefault ) );
        gpuErrchk( hipHostAlloc( (void**) &B , N * sizeof(*B) ,hipHostMallocDefault ) );
        gpuErrchk( hipHostAlloc( (void**) &C1 , N * sizeof(*C1) ,hipHostMallocDefault ) );
        gpuErrchk( hipHostAlloc( (void**) &C2 , N * sizeof(*C2) ,hipHostMallocDefault ) );
        
        for ( int i = 0; i < N; i++ ) {
                A[ i ] = 1;
                B[ i ] = 2;
                C1[i] = 0;
                C2[i] = 0;
        }
        
        /************************************
                  STREAM CREATION
        ************************************/
        hipStream_t Stream1[ NbStreams ];
        for ( int i = 0; i < NbStreams; i++ )
        	gpuErrchk( hipStreamCreate(&Stream1[ i ]) );
        
        hipStream_t Stream2[ NbStreams ];
        for ( int i = 0; i < NbStreams; i++ )
        	gpuErrchk( hipStreamCreate(&Stream2[ i ]) );

        const int StreamSize = N / NbStreams;
        
        /************************************
               MANAGED DEVICE ALLOCATION
        ************************************/
        float *d_A, *d_B, *d_C;
        gpuErrchk( hipMallocManaged(  &d_A , N * sizeof(*d_A)) );
        gpuErrchk( hipMallocManaged(  &d_B , N * sizeof(*d_B)) );
        gpuErrchk( hipMallocManaged(  &d_C , N * sizeof(*d_C)) );
        
        gpuErrchk( hipMemPrefetchAsync(  d_B , N * sizeof(*d_B), 0) );
        gpuErrchk( hipMemPrefetchAsync(  d_A , N * sizeof(*d_A), 0) );
        gpuErrchk( hipMemPrefetchAsync(  d_C , N * sizeof(*d_C), 0) );
        
        /************************************
                 EXECUTION ON MANAGED
        ************************************/
        for ( int i = 0; i < NbStreams; i++ )
        {
                int Offset = i * StreamSize;

                gpuErrchk( hipMemcpyAsync(&d_A[ Offset ], &A[ Offset ], StreamSize * sizeof(*A), hipMemcpyHostToDevice, Stream2[ i ]) );
                gpuErrchk( hipMemcpyAsync(&d_B[ Offset ], &B[ Offset ], StreamSize * sizeof(*B), hipMemcpyHostToDevice, Stream2[ i ]) );

                Add<<< StreamSize / Threads, Threads, 0, Stream2[ i ]>>>( Offset+StreamSize ,Offset, d_A , d_B , d_C );

                gpuErrchk( hipMemcpyAsync(&C2[ Offset ], &d_C[ Offset ], StreamSize * sizeof(*d_C), hipMemcpyDeviceToHost, Stream2[ i ]) );

        }
        
        /************************************
               STATIC DEVICE ALLOCATION
        ************************************/
        float *devA , *devB , *devC;
        gpuErrchk( hipMalloc( (void**) &devA , N * sizeof(*devA)) );
        gpuErrchk( hipMalloc( (void**) &devB , N * sizeof(*devB)) );
        gpuErrchk( hipMalloc( (void**) &devC , N * sizeof(*devC)) );
  
        /************************************
                     EXECUTION
        ************************************/
        for ( int i = 0; i < NbStreams; i++ )
        {
                int Offset = i * StreamSize;

                gpuErrchk( hipMemcpyAsync(&devA[ Offset ], &A[ Offset ], StreamSize * sizeof(*A), hipMemcpyHostToDevice, Stream1[ i ]) );
                gpuErrchk( hipMemcpyAsync(&devB[ Offset ], &B[ Offset ], StreamSize * sizeof(*B), hipMemcpyHostToDevice, Stream1[ i ]) );

                Add<<< StreamSize / Threads, Threads, 0, Stream1[i]>>>( Offset+StreamSize ,Offset, devA , devB , devC );

                gpuErrchk( hipMemcpyAsync(&C1[ Offset ], &devC[ Offset ], StreamSize * sizeof(*devC), hipMemcpyDeviceToHost, Stream1[ i ]) );

        }
        
        hipDeviceSynchronize();
        
        /************************************
                    RESULT CHECK
        ************************************/
        for ( int i = 0; i < N; i++ ) {
                if (C1[i] != (A[i]+B[i])) {
                printf("mismatch at %d, was: %f, should be: %f (first)\n", i, C1[i], (A[i]+B[i])); return 1;
        	}
        }
        
                for ( int i = 0; i < N; i++ ) {
                if (C2[i] != (A[i]+B[i])) {
                printf("mismatch at %d, was: %f, should be: %f (second)\n", i, C2[i], (A[i]+B[i])); return 1;
        	}
        }

        // DESTROY CONTEXT
        for ( int i = 0; i < NbStreams; i++ )
                gpuErrchk( hipStreamDestroy(Stream1[ i ]) );
                for ( int i = 0; i < NbStreams; i++ )
                gpuErrchk( hipStreamDestroy(Stream2[ i ]) );

        gpuErrchk( hipFree(devA) );
        gpuErrchk( hipFree(devB) );
        gpuErrchk( hipFree(devC) );

        gpuErrchk( hipHostFree(A) );
        gpuErrchk( hipHostFree(B) );
        gpuErrchk( hipHostFree(C1) );

        return 0;

}
