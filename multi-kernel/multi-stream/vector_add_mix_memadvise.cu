#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#define gpuErrchk(ans) { gpuAssert( (ans), __FILE__, __LINE__ ); }

inline void
gpuAssert( hipError_t code, const char * file, int line, bool abort = true )
{
        if ( hipSuccess != code )
        {
                fprintf( stderr, "\nGPUassert: %s %s %d\n", hipGetErrorString( code ), file, line );
                if ( abort )
                        exit( code );
        }

return;

} /* gpuAssert */

__global__ void Add( int N ,int Offset ,float * devA , float * devB , float *devC )
{

        for ( int idx = blockIdx.x * blockDim.x + threadIdx.x + Offset; idx < N; idx += blockDim.x * gridDim.x )

                devC[ idx ] = devA[ idx ] + devB[ idx ];

}

int main() {

        size_t free_t, total_t;

        gpuErrchk( hipMemGetInfo(&free_t, &total_t) );
        
        unsigned long N = ((0.3 * free_t) / sizeof(float));
        
        printf("allocating %lu bytes per array\n", N*4);
        
        int Threads = 1024;

        const int NbStreams = 8;
        
        /************************************
                  HOST ALLOCATION
        ************************************/
        float *A , *B , *C1, *C2;
        gpuErrchk( hipHostMalloc( (void**) &A , N * sizeof(*A)) );
        gpuErrchk( hipHostMalloc( (void**) &B , N * sizeof(*B)) );
        gpuErrchk( hipHostMalloc( (void**) &C1 , N * sizeof(*C1)) );
        gpuErrchk( hipHostMalloc( (void**) &C2 , N * sizeof(*C2)) );
        
        /************************************
                  STREAM CREATION
        ************************************/
        hipStream_t Stream1[ NbStreams ];
        for ( int i = 0; i < NbStreams; i++ )
        	gpuErrchk( hipStreamCreate(&Stream1[ i ]) );
        
        hipStream_t Stream2[ NbStreams ];
        for ( int i = 0; i < NbStreams; i++ )
        	gpuErrchk( hipStreamCreate(&Stream2[ i ]) );

        const int StreamSize = N / NbStreams;
        
        /************************************
               MANAGED DEVICE ALLOCATION
        ************************************/
        float *d_A, *d_B, *d_C;
        gpuErrchk( hipMallocManaged(  &d_A , N * sizeof(*d_A)) );
        gpuErrchk( hipMallocManaged(  &d_B , N * sizeof(*d_B)) );
        gpuErrchk( hipMallocManaged(  &d_C , N * sizeof(*d_C)) );
  
        gpuErrchk( hipMemAdvise(d_A, N * sizeof(*d_A), hipMemAdviseSetPreferredLocation, -1) );
        gpuErrchk( hipMemAdvise(d_B, N * sizeof(*d_A), hipMemAdviseSetPreferredLocation, -1) );
        gpuErrchk( hipMemAdvise(d_C, N * sizeof(*d_A), hipMemAdviseSetPreferredLocation, -1) );
        
        gpuErrchk( hipMemAdvise(d_A, N * sizeof(*d_A), hipMemAdviseSetAccessedBy, 0) );
        gpuErrchk( hipMemAdvise(d_B, N * sizeof(*d_A), hipMemAdviseSetAccessedBy, 0) );
        gpuErrchk( hipMemAdvise(d_C, N * sizeof(*d_A), hipMemAdviseSetAccessedBy, 0) );
  
        /************************************
               STATIC DEVICE ALLOCATION
        ************************************/
        float *devA , *devB , *devC;
        gpuErrchk( hipMalloc( (void**) &devA , N * sizeof(*devA)) );
        gpuErrchk( hipMalloc( (void**) &devB , N * sizeof(*devB)) );
        gpuErrchk( hipMalloc( (void**) &devC , N * sizeof(*devC)) );
        
        /************************************
                 DATA INITIALIZATION
        ************************************/
        for ( int i = 0; i < N; i++ ) {
                A[ i ] = 1;
                B[ i ] = 2;
                C1[i] = 0;
                C2[i] = 0;
                d_A[i] = 1;
                d_B[i] = 2;
                d_C[i] = 0;
        }
        /************************************
                 EXECUTION ON STATIC
        ************************************/
        for ( int i = 0; i < NbStreams; i++ )
        {
                int Offset = i * StreamSize;

                gpuErrchk( hipMemcpyAsync(&devA[ Offset ], &A[ Offset ], StreamSize * sizeof(*A), hipMemcpyHostToDevice, Stream1[ i ]) );
                gpuErrchk( hipMemcpyAsync(&devB[ Offset ], &B[ Offset ], StreamSize * sizeof(*B), hipMemcpyHostToDevice, Stream1[ i ]) );

                Add<<< StreamSize / Threads, Threads, 0, Stream1[i]>>>( Offset+StreamSize ,Offset, devA , devB , devC );

                gpuErrchk( hipMemcpyAsync(&C1[ Offset ], &devC[ Offset ], StreamSize * sizeof(*devC), hipMemcpyDeviceToHost, Stream1[ i ]) );

        }
        
        /************************************
                 EXECUTION ON MANAGED
        ************************************/
        for ( int i = 0; i < NbStreams; i++ )
        {
                int Offset = i * StreamSize;

                //gpuErrchk( hipMemcpyAsync(&d_A[ Offset ], &A[ Offset ], StreamSize * sizeof(*A), hipMemcpyHostToDevice, Stream2[ i ]) );
                //gpuErrchk( hipMemcpyAsync(&d_B[ Offset ], &B[ Offset ], StreamSize * sizeof(*B), hipMemcpyHostToDevice, Stream2[ i ]) );

                gpuErrchk( hipMemPrefetchAsync( &d_A[ Offset ] , StreamSize * sizeof(*A), -1) );  
                gpuErrchk( hipMemPrefetchAsync( &d_B[ Offset ] , StreamSize * sizeof(*B), -1) );  
              
                Add<<< StreamSize / Threads, Threads, 0, Stream2[i]>>>( Offset+StreamSize ,Offset, d_A , d_B , d_C );
          
                gpuErrchk( hipMemPrefetchAsync( &d_C[Offset] , StreamSize * sizeof(*d_C), -1) );
               
                //gpuErrchk( hipMemcpyAsync(&C2[ Offset ], &d_C[Offset], StreamSize * sizeof(*d_C), hipMemcpyDeviceToHost, Stream2[ i ]) );

        }
        
        hipDeviceSynchronize();
        
        /************************************
                    RESULT CHECK
        ************************************/
        for ( int i = 0; i < N-6; i++ ) {
                if (C1[i] != (A[i]+B[i])) {
                	printf("mismatch at %d, was: %f, should be: %f (first)\n", i, C1[i], (A[i]+B[i])); return 1;
        	}
        }
        
        for ( int i = 0; i < N-6; i++ ) {
                if (C2[i] != (A[i]+B[i])) {
                	printf("mismatch at %d, was: %f, should be: %f (second)\n", i, C2[i], (A[i]+B[i])); return 1;
        	}
        }
  
        printf("no errors\n");

        // DESTROY CONTEXT
        for ( int i = 0; i < NbStreams; i++ )
                gpuErrchk( hipStreamDestroy(Stream1[ i ]) );
        for ( int i = 0; i < NbStreams; i++ )
                gpuErrchk( hipStreamDestroy(Stream2[ i ]) );

        gpuErrchk( hipFree(devA) );
        gpuErrchk( hipFree(devB) );
        gpuErrchk( hipFree(devC) );

        gpuErrchk( hipHostFree(A) );
        gpuErrchk( hipHostFree(B) );
        gpuErrchk( hipHostFree(C1) );

        return 0;

}
