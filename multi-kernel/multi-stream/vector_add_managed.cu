
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#define gpuErrchk(ans) { gpuAssert( (ans), __FILE__, __LINE__ ); }

inline void
gpuAssert( hipError_t code, const char * file, int line, bool abort = true )
{
        if ( hipSuccess != code )
        {
                fprintf( stderr, "\nGPUassert: %s %s %d\n", hipGetErrorString( code ), file, line );
                if ( abort )
                        exit( code );
        }

return;

} /* gpuAssert */

__global__ void Add( int N ,int Offset ,float * devA , float * devB , float *devC )
{

        for ( int idx = blockIdx.x * blockDim.x + threadIdx.x + Offset; idx < N; idx += blockDim.x * gridDim.x )

                devC[ idx ] = devA[ idx ] + devB[ idx ];

}

int main()
{

        //int N = 4000000;
        unsigned long N = 1395864368;  //c.a. 1,3 * 4 GB
        //unsigned long N = 1288490188;  //c.a. 1,2 * 4 GB
        
        int Threads = 256;

        const int NbStreams = 8;
        
        /************************************
                  HOST ALLOCATION
        ************************************/
        float *A , *B , *C1, *C2;
        gpuErrchk( hipHostAlloc( (void**) &A , N * sizeof(*A) ,hipHostMallocDefault ) );
        gpuErrchk( hipHostAlloc( (void**) &B , N * sizeof(*B) ,hipHostMallocDefault ) );
        gpuErrchk( hipHostAlloc( (void**) &C1 , N * sizeof(*C1) ,hipHostMallocDefault ) );
        gpuErrchk( hipHostAlloc( (void**) &C2 , N * sizeof(*C2) ,hipHostMallocDefault ) );
        
        for ( int i = 0; i < N; i++ ) {
                A[ i ] = 1;
                B[ i ] = 2;
                C1[i] = 0;
                C2[i] = 0;
        }
        
        /************************************
               MANAGED DEVICE ALLOCATION
        ************************************/
        float *devA , *devB , *devC;
        gpuErrchk( hipMallocManaged(  &devA , N * sizeof(*devA)) );
        gpuErrchk( hipMallocManaged(  &devB , N * sizeof(*devB)) );
        gpuErrchk( hipMallocManaged(  &devC , N * sizeof(*devC)) );
        
        float *d_A, *d_B, *d_C;
        gpuErrchk( hipMallocManaged(  &d_A , N * sizeof(*devA)) );
        gpuErrchk( hipMallocManaged(  &d_B , N * sizeof(*devB)) );
        gpuErrchk( hipMallocManaged(  &d_C , N * sizeof(*devC)) );
    	
    	// STREAM CREATION
        hipStream_t Stream1[ NbStreams ];
        for ( int i = 0; i < NbStreams; i++ )
        	gpuErrchk( hipStreamCreate(&Stream1[ i ]) );
        
        hipStream_t Stream2[ NbStreams ];
        for ( int i = 0; i < NbStreams; i++ )
        	gpuErrchk( hipStreamCreate(&Stream2[ i ]) );

        const int StreamSize = N / NbStreams;
        dim3 block(1024);
        dim3 grid(((N/StreamSize)+block.x-1)/block.x);

        /************************************
                     EXECUTION
        ************************************/
        for ( int i = 0; i < NbStreams; i++ )
        {
                int Offset = i * StreamSize;

                gpuErrchk( hipMemcpyAsync(&devA[ Offset ], &A[ Offset ], StreamSize * sizeof(*A), hipMemcpyHostToDevice, Stream1[ i ]) );
                gpuErrchk( hipMemcpyAsync(&devB[ Offset ], &B[ Offset ], StreamSize * sizeof(*B), hipMemcpyHostToDevice, Stream1[ i ]) );

                Add<<< StreamSize / Threads, Threads, 0, Stream1[i]>>>( Offset+StreamSize ,Offset, devA , devB , devC );

                gpuErrchk( hipMemcpyAsync(&C1[ Offset ], &devC[ Offset ], StreamSize * sizeof(*devC), hipMemcpyDeviceToHost, Stream1[ i ]) );

        }
        
        for ( int i = 0; i < NbStreams; i++ )
        {
                int Offset = i * StreamSize;

                gpuErrchk( hipMemcpyAsync(&d_A[ Offset ], &A[ Offset ], StreamSize * sizeof(*A), hipMemcpyHostToDevice, Stream2[ i ]) );
                gpuErrchk( hipMemcpyAsync(&d_B[ Offset ], &B[ Offset ], StreamSize * sizeof(*B), hipMemcpyHostToDevice, Stream2[ i ]) );

                Add<<< StreamSize / Threads, Threads, 0, Stream2[ i ]>>>( Offset+StreamSize ,Offset, d_A , d_B , d_C );

                gpuErrchk( hipMemcpyAsync(&C2[ Offset ], &d_C[ Offset ], StreamSize * sizeof(*d_C), hipMemcpyDeviceToHost, Stream2[ i ]) );

        }
        
        hipDeviceSynchronize();
        
        /************************************
                    RESULT CHECK
        ************************************/
        for ( int i = 0; i < N; i++ ) {
                if (C1[i] != (A[i]+B[i])) {
                printf("mismatch at %d, was: %f, should be: %f (static)\n", i, C1[i], (A[i]+B[i])); return 1;
        	}
        	if (C2[i] != (A[i]+B[i])) {
                printf("mismatch at %d, was: %f, should be: %f (managed)\n", i, C2[i], (A[i]+B[i])); return 2;
        	}
        }

        // DESTROY CONTEXT
        for ( int i = 0; i < NbStreams; i++ )
                gpuErrchk( hipStreamDestroy(Stream1[ i ]) );
                for ( int i = 0; i < NbStreams; i++ )
                gpuErrchk( hipStreamDestroy(Stream2[ i ]) );

        gpuErrchk( hipFree(devA) );
        gpuErrchk( hipFree(devB) );
        gpuErrchk( hipFree(devC) );

        gpuErrchk( hipHostFree(A) );
        gpuErrchk( hipHostFree(B) );
        gpuErrchk( hipHostFree(C1) );

        return 0;

}
