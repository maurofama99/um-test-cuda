
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#define gpuErrchk(ans) { gpuAssert( (ans), __FILE__, __LINE__ ); }

inline void
gpuAssert( hipError_t code, const char * file, int line, bool abort = true )
{
        if ( hipSuccess != code )
        {
                fprintf( stderr, "\nGPUassert: %s %s %d\n", hipGetErrorString( code ), file, line );
                if ( abort )
                        exit( code );
        }

return;

} /* gpuAssert */

__global__ void Add( int N ,int Offset ,float * devA , float * devB , float *devC )
{

        for ( int idx = blockIdx.x * blockDim.x + threadIdx.x + Offset; idx < N; idx += blockDim.x * gridDim.x )

                devC[ idx ] = devA[ idx ] + devB[ idx ];

}

int main()
{

        int N = 4000000;

        int Threads = 256;

        const int NbStreams = 8;

        float *A , *B , *C;
        gpuErrchk( hipHostAlloc( (void**) &A , N * sizeof(*A) ,hipHostMallocDefault ) );
        gpuErrchk( hipHostAlloc( (void**) &B , N * sizeof(*B) ,hipHostMallocDefault ) );
        gpuErrchk( hipHostAlloc( (void**) &C , N * sizeof(*C) ,hipHostMallocDefault ) );

        for ( int i = 0; i < N; i++ )
        {
                A[ i ] = i;
                B[ i ] = i + 1;
        }

        float *devA , *devB , *devC;
        gpuErrchk( hipMalloc( (void**) &devA , N * sizeof(*devA)) );
        gpuErrchk( hipMalloc( (void**) &devB , N * sizeof(*devB)) );
        gpuErrchk( hipMalloc( (void**) &devC , N * sizeof(*devC)) );

        hipEvent_t EventPre,
                    EventPost;
        float PostPreTime;

        gpuErrchk( hipEventCreate( &EventPre ) );
        gpuErrchk( hipEventCreate( &EventPost ) );

    hipStream_t Stream[ NbStreams ];
        for ( int i = 0; i < NbStreams; i++ )
        gpuErrchk( hipStreamCreate(&Stream[ i ]) );

#ifdef NOSTREAMS

        gpuErrchk( cudaEventRecord(EventPre ) );

        gpuErrchk( cudaMemcpy(devA, A, N * sizeof(*A), cudaMemcpyHostToDevice) );
        gpuErrchk( cudaMemcpy(devB, B, N * sizeof(*B), cudaMemcpyHostToDevice) );
//        gpuErrchk( cudaMemcpy(devC, C, N * sizeof(*C), cudaMemcpyHostToDevice) );

        Add<<< N / Threads, Threads>>>( N ,0, devA , devB , devC );

//        gpuErrchk( cudaMemcpy(A, devA, N * sizeof(*A), cudaMemcpyDeviceToHost) );
//        gpuErrchk( cudaMemcpy(B, devB, N * sizeof(*B), cudaMemcpyDeviceToHost) );
        gpuErrchk( cudaMemcpy(C, devC, N * sizeof(*C), cudaMemcpyDeviceToHost) );

        gpuErrchk( cudaEventRecord( EventPost ) );
        gpuErrchk( cudaEventSynchronize( EventPost ) );
        gpuErrchk( cudaEventElapsedTime( &PostPreTime, EventPre, EventPost ) );
        printf( "\nTime not using streams: %f ms\n", PostPreTime );

#else

        const int StreamSize = N / NbStreams;
        gpuErrchk( hipEventRecord( EventPre ) );
        for ( int i = 0; i < NbStreams; i++ )
        {
                int Offset = i * StreamSize;

                gpuErrchk( hipMemcpyAsync(&devA[ Offset ], &A[ Offset ], StreamSize * sizeof(*A), hipMemcpyHostToDevice, Stream[ i ]) );
                gpuErrchk( hipMemcpyAsync(&devB[ Offset ], &B[ Offset ], StreamSize * sizeof(*B), hipMemcpyHostToDevice, Stream[ i ]) );
//                gpuErrchk( cudaMemcpyAsync(&devC[ Offset ], &C[ Offset ], StreamSize * sizeof(*C), cudaMemcpyHostToDevice, Stream[ i ]) );

                Add<<< StreamSize / Threads, Threads, 0, Stream[ i ]>>>( Offset+StreamSize ,Offset, devA , devB , devC );

//                gpuErrchk( cudaMemcpyAsync(&A[ Offset ], &devA[ Offset ], StreamSize * sizeof(*devA), cudaMemcpyDeviceToHost, Stream[ i ]) );
//                gpuErrchk( cudaMemcpyAsync(&B[ Offset ], &devB[ Offset ], StreamSize * sizeof(*devB), cudaMemcpyDeviceToHost, Stream[ i ]) );
                gpuErrchk( hipMemcpyAsync(&C[ Offset ], &devC[ Offset ], StreamSize * sizeof(*devC), hipMemcpyDeviceToHost, Stream[ i ]) );

        }

        gpuErrchk( hipEventRecord( EventPost ) );
        gpuErrchk( hipEventSynchronize( EventPost ) );
        gpuErrchk( hipEventElapsedTime( &PostPreTime, EventPre,EventPost ) );
        printf( "\nTime using streams: %f ms\n", PostPreTime );

#endif /* ! USE_STREAMS */

        for ( int i = 0; i < N; i++ )
                if (C[i] != (A[i]+B[i])) {printf("mismatch at %d, was: %f, should be: %f\n", i, C[i], (A[i]+B[i])); return 1;}

        for ( int i = 0; i < NbStreams; i++ )
                gpuErrchk( hipStreamDestroy(Stream[ i ]) );

        gpuErrchk( hipFree(devA) );
        gpuErrchk( hipFree(devB) );
        gpuErrchk( hipFree(devC) );

        gpuErrchk( hipHostFree(A) );
        gpuErrchk( hipHostFree(B) );
        gpuErrchk( hipHostFree(C) );

        gpuErrchk( hipEventDestroy(EventPre) );
    gpuErrchk( hipEventDestroy(EventPost) );

        printf("\n");

        return 0;

}
