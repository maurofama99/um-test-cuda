#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
/*
#define CHECK(call)                                                    \
{                                                                      \
    const cudaError_t error = call;                                    \
    if (error != cudaSuccess)                                          \
    {                                                                  \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);         \
        fprintf(stderr, "code: %d, reason: %s\n", error,               \
        cudaGetErrorString(error));                                    \
        exit(1);                                                       \
    }                                                                  \
}
*/
inline void CHECK(const hipError_t error)
{
    if(error != hipSuccess)
    {
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);
        fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error));
        exit(1);
    }
}

double cpuTimer()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

void initialData(float *arr, int size)
{
    time_t t;
    srand((unsigned)time(&t));  // seed
    for(int i=0;i<size;i++)
                arr[i]=1;
}
void AddVecOnHost(float *A, float *B, float *C, const int size)
{
#pragma omp parallel for
    for(int i=0;i<size;i++)
        C[i] = A[i] + B[i];
}

__global__ void AddVecOnGPU(float *A, float *B, float *C, const int size)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx<size) C[idx] = A[idx] + B[idx];
}

void checkResult(float *host, float *gpu, const int N)
{
    double epsilon = 1.0e-8;
    bool match = 1;
    for(int i=0;i<N;i++)
    {
        if(abs(host[i] - gpu[i]) > epsilon)
        {
            match = 0;
            printf("Vector do not match!\n");
            printf("host %5.2f, gpu %5.2f at current %d\n", host[i], gpu[i], i);
            break;
        }
    }
    if(match) printf("Vectors match.\n");
}
int main(int argc, char **argv)
{
    //unsigned long nSize = 1UL<<31;   //16M
    unsigned long nSize = 2147483648;  //c.a. 2 * 4 GB
    //unsigned long nSize = 1288490188; //c.a. 1,2 * 4 GB
    printf("Vector size : %ld\n", nSize);
/*********** on HOST *******************/
    // malloc host memory
    size_t nBytes = nSize*sizeof(float);

    float *h_A, *h_B, *hostResult, *gpuResult1, *gpuResult2;
    h_A = (float*)malloc(nBytes);
    h_B = (float*)malloc(nBytes);
    hostResult = (float*)malloc(nBytes);
    gpuResult1 = (float*)malloc(nBytes);

    double iStart, iEnd;
    double ElapsedTime;

    initialData(h_A, nSize);
    initialData(h_B, nSize);

    memset(hostResult, 0, nBytes);
    memset(gpuResult1, 0, nBytes);

    iStart=cpuTimer();
    AddVecOnHost(h_A, h_B, hostResult, nSize);
    iEnd = cpuTimer();
    ElapsedTime = iEnd - iStart;
    printf("Elapsed Time in AddVecOnHost : %f\n",ElapsedTime);
/*****************************************/

/********** ON GPU **********************/
    // malloc device global memory
    float *d_A, *d_B, *d_C;

    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));

    CHECK(hipMallocManaged(&d_C, nBytes));

    // Data transfer : Host --> Device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));

    // dimension of thread block and grid
    dim3 block(1024);
    dim3 grid((nSize+block.x-1)/block.x);

    // create tow events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    iStart = cpuTimer();
    float Etime;
    hipEventRecord(start);
    AddVecOnGPU<<<grid, block>>>(d_A, d_B, d_C, nSize);	    
    CHECK(hipDeviceSynchronize());
//  ElapsedTime = cpuTimer() - iStart;
    hipEventRecord(stop);
    ElapsedTime = cpuTimer() - iStart;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&Etime, start, stop);
    printf("Elapsed Time in AddVecOnGPU<<<%d, %d>>> : %f ms\n", grid.x, block.x, Etime);
//  printf("GPU Timer : %f ms , CPU Timer : %f ms\n",Etime, ElapsedTime*1000.0);

    CHECK(hipMemcpy(gpuResult1, d_C, nBytes, hipMemcpyDeviceToHost));
/****************************************/

    // check results
    checkResult(hostResult, gpuResult1, nSize);

    // memory deallocate
    free(h_A),      free(h_B),      free(hostResult),       free(gpuResult1);
    CHECK(hipFree(d_A)),   CHECK(hipFree(d_B)),   CHECK(hipFree(d_C));

    return 0;
}
